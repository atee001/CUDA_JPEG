#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"

int main (int argc, char *argv[])
{

    // Timer timer;
    hipError_t cuda_ret;
    // Initialize host variables ----------------------------------------------
    cv::Mat image = cv::imread("/home/eemaj/atee/ee147/jpeg/CUDA_JPEG/images/lena_std.tif");    
    if (image.empty())
    {
        printf("Failed to read image exitting...");
        return 1;
    }

    //preprocess convert image to 512 x 512 and single channel

    cv::resize(image, image, cv::Size(512, 512));
    cv::cvtColor(image, image, cv::COLOR_BGR2GRAY);    

    printf("\nSetting up the problem..."); fflush(stdout);
    // startTime(&timer);

    // float *A_h, *B_h, *C_h;
    // float *A_d, *B_d, *C_d;
    // size_t mat_sz;
    // unsigned matDim;
    // dim3 dim_grid, dim_block;
    size_t imageSize = image.rows*image.cols*sizeof(float);

    if (argc == 1) {
        imageSize = image.rows*image.cols*sizeof(float);
    } 
    else {
        printf("\n    Invalid input parameters!"
      "\n    Usage: ./mat-add                # All Images are 512 x 512"
      "\n");
        exit(0);
    }
    cv::Mat image_float; 
    image.convertTo(image_float, CV_32F);
    float *d_image, *DCT_res, *IDCT_res;
   
    hipMalloc((void**)&d_image, imageSize);
    hipMalloc((void**)&DCT_res, imageSize);
    hipMalloc((void**)&IDCT_res, imageSize);

    hipDeviceSynchronize();    

    hipMemcpy(d_image, image_float.ptr<float>(), imageSize, hipMemcpyHostToDevice);

    printf("Testing");
    
    hipDeviceSynchronize();  
    LaunchDCT(image.rows, image.cols, d_image, DCT_res);
    hipDeviceSynchronize();
    LaunchIDCT(image.rows, image.cols, DCT_res, IDCT_res);

    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");

    float* h_outputImage = (float*)malloc(imageSize*sizeof(float));
    hipMemcpy(h_outputImage, IDCT_res, imageSize, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();   

    uint8_t* outputImage = (uint8_t*)malloc(image.rows*image.cols*sizeof(uint8_t));

    for(unsigned int i = 0; i < image.rows*image.cols; i++){
        outputImage[i] = static_cast<uint8_t>(h_outputImage[i]);
    }

    cv::namedWindow("Image Window", cv::WINDOW_NORMAL);
    cv::imshow("Image Window", image);

    cv::Mat resultImage(image.rows, image.cols, CV_8UC1, outputImage);
    cv::imshow("Resultant Image", resultImage);
    cv::namedWindow("Resultant Image", cv::WINDOW_NORMAL);
    cv::waitKey(0);
    

    free(h_outputImage);
    free(outputImage);
    hipFree(d_image);
    hipFree(DCT_res);
    hipFree(IDCT_res);


    cv::destroyWindow("Image Window");
    cv::destroyWindow("Resultant Image");
    /*************************************************************************/
    return 0;
}

