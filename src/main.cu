#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"

double calculateMSE(const cv::Mat& image1, const cv::Mat& image2) {

    cv::Mat diff;
    cv::absdiff(image1, image2, diff); //absolute value difference    
    cv::Mat squaredDiff = diff.mul(diff); // Calculate the squared difference    
    cv::Scalar mse = cv::mean(squaredDiff); // Calculate the mean squared error
    double mseValue = mse[0];  // single channel scaler since grayscale image
    return mseValue/(image1.rows*image1.cols);
}

cv::Mat createZonalFilter15()
{
    cv::Mat zonalFilter = cv::Mat::zeros(8, 8, CV_64F);
    double maskData[8][8] = {
        1, 1, 1, 1, 1, 0, 0, 0,
        1, 1, 1, 1, 0, 0, 0, 0,
        1, 1, 1, 0, 0, 0, 0, 0,
        1, 1, 0, 0, 0, 0, 0, 0,
        1, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0
    };
    memcpy(zonalFilter.data, maskData, sizeof(maskData));
    return zonalFilter;
}

cv::Mat createZonalFilter32()
{
    cv::Mat zonalFilter = cv::Mat::zeros(8, 8, CV_64F);
    double maskData[8][8] = {
        1, 1, 1, 1, 1, 1, 1, 1,
        1, 1, 1, 1, 1, 1, 1, 1,
        1, 1, 1, 1, 1, 1, 0, 0,
        1, 1, 1, 1, 0, 0, 0, 0,
        1, 1, 1, 0, 0, 0, 0, 0,
        1, 1, 0, 0, 0, 0, 0, 0,
        1, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0
    };
    memcpy(zonalFilter.data, maskData, sizeof(maskData));
    return zonalFilter;
}

cv::Mat createZonalFilterAll()
{
    cv::Mat zonalFilter = cv::Mat::zeros(8, 8, CV_64F);
    double maskData[8][8] = {
        1, 1, 1, 1, 1, 1, 1, 1,
        1, 1, 1, 1, 1, 1, 1, 1,
        1, 1, 1, 1, 1, 1, 1, 1,
        1, 1, 1, 1, 1, 1, 1, 1,
        1, 1, 1, 1, 1, 1, 1, 1,
        1, 1, 1, 1, 1, 1, 1, 1,
        1, 1, 1, 1, 1, 1, 1, 1,
        1, 1, 1, 1, 1, 1, 1, 1
    };
    memcpy(zonalFilter.data, maskData, sizeof(maskData));
    return zonalFilter;
}

int main (int argc, char *argv[])
{
    int choice;
    
    printf("Choose the zonal filter:\n");
    printf("1. Retain 15 coefficients\n");
    printf("2. Retain 32 coefficients\n");
    printf("3. Retain all coefficients\n");
    printf("Enter your choice (1-3): ");
    scanf("%d", &choice);

    cv::Mat zonalFilter;

    switch (choice){
    case 1:
        zonalFilter = createZonalFilter15();
        break;
    case 2:
        zonalFilter = createZonalFilter32();
        break;
    case 3:
        zonalFilter = createZonalFilterAll();
        break;
    default:
        printf("Invalid choice!\n");
        return -1;
    }

    printf("Selected Zonal Filter:\n");
    for (int i = 0; i < 8; i++)
    {
        for (int j = 0; j < 8; j++)
        {
            printf("%.0f ", zonalFilter.at<double>(i, j));
        }
        printf("\n");
    }

    // Timer timer;
    hipError_t cuda_ret;
    // Initialize host variables ----------------------------------------------
    cv::Mat image = cv::imread("/home/eemaj/atee/ee147/jpeg/CUDA_JPEG/images/lena_std.tif");    
    if (image.empty())
    {
        printf("Failed to read image exitting...");
        return 1;
    }

    //preprocess convert image to 512 x 512 and single channel

    cv::resize(image, image, cv::Size(512, 512));
    cv::cvtColor(image, image, cv::COLOR_BGR2GRAY);    

    printf("\nSetting up the problem..."); fflush(stdout);
    // startTime(&timer);

    // double *A_h, *B_h, *C_h;
    // double *A_d, *B_d, *C_d;
    // size_t mat_sz;
    // unsigned matDim;
    // dim3 dim_grid, dim_block;
    size_t imageSize = image.rows*image.cols*sizeof(double);

    if (argc == 1) {
        imageSize = image.rows*image.cols*sizeof(double);
    } 
    else {
        printf("\n    Invalid input parameters!"
      "\n    Usage: ./mat-add                # All Images are 512 x 512"
      "\n");
        exit(0);
    }
    cv::Mat image_double; 
    image.convertTo(image_double, CV_64F);
    double *d_image, *f_image, *r_image;
   
    hipMalloc((void**)&d_image, imageSize);
    hipMalloc((void**)&f_image, imageSize);
    hipMalloc((void**)&r_image, imageSize);

    hipDeviceSynchronize();    

    hipMemcpy(d_image, image_double.ptr<double>(), imageSize, hipMemcpyHostToDevice);

    printf("Testing");
    
    hipDeviceSynchronize();  

    compress(image.rows, image.cols, d_image, f_image); //returns image in frequency domain
    hipDeviceSynchronize();
    decompress(image.rows, image.cols, f_image, r_image); //returns image in spatial domain
    // hipDeviceSynchronize();
    // LaunchIDCT(image.rows, image.cols, IDCT_res, temp);
    // hipDeviceSynchronize();
    // LaunchDCT(image.rows, image.cols, temp, result);

    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");

    double* h_outputImage = (double*)malloc(imageSize);
    double* f_outputImage = (double*)malloc(imageSize);
    hipMemcpy(h_outputImage, r_image, imageSize, hipMemcpyDeviceToHost);    
    hipMemcpy(f_outputImage, f_image, imageSize, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();   

    // for (unsigned int i = 0; i < image.rows * image.cols; i++) {
    //     h_outputImage[i] *= 255.0;
    // }
    

    // Convert the matrix to CV_8U data type


    cv::Mat resultImage(image.rows, image.cols, CV_64F);
    memcpy(resultImage.data, h_outputImage, imageSize);
    cv::normalize(resultImage, resultImage, 0, 255, cv::NORM_MINMAX, CV_8U);
    resultImage.convertTo(resultImage, CV_8U);
    // for (int i = 0; i < resultImage.rows; i++) {
    //     for (int j = 0; j < resultImage.cols; j++) {
    //         resultImage.at<uint8_t>(i, j) = static_cast<uint8_t>(h_outputImage[i * resultImage.cols + j]);
    //     }
    // }

    cv::Mat frequencyImage(image.rows, image.cols, CV_64F);
    
    memcpy(frequencyImage.data, f_outputImage, imageSize);

    cv::log(cv::abs(frequencyImage) + 1, frequencyImage);
    cv::normalize(frequencyImage, frequencyImage, 0, 255, cv::NORM_MINMAX, CV_8U);
    // frequencyImage.convertTo(frequencyImage, CV_8U);
    cv::namedWindow("Frequency Image", cv::WINDOW_NORMAL);
    cv::imshow("Frequency Image", frequencyImage);    
    // cv::normalize(resultImage, resultImage, 0, 255, cv::NORM_MINMAX, CV_8U);
    

    cv::namedWindow("Original Image", cv::WINDOW_NORMAL);
    cv::imshow("Original Image", image);
    cv::namedWindow("Decompressed Image", cv::WINDOW_NORMAL);
    cv::imshow("Decompressed Image", resultImage);   
    cv::waitKey(0);
    

    free(h_outputImage);
    free(f_outputImage);
    // free(outputImage);
    hipFree(d_image);
    hipFree(f_image);
    hipFree(r_image);


    // cv::destroyWindow("Image Window");
    cv::destroyWindow("Decompressed Image");
    cv::destroyWindow("Original Image");
    cv::destroyWindow("Frequency Image");

    double mse = calculateMSE(image, resultImage);
    std::cout << "MSE of Original Image Vs Decompressed Image: " << mse << std::endl;



    /*************************************************************************/
    return 0;
}

