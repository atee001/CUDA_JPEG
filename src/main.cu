#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"

int main (int argc, char *argv[])
{

    // Timer timer;
    hipError_t cuda_ret;
    // Initialize host variables ----------------------------------------------
    cv::Mat image = cv::imread("/home/eemaj/atee/ee147/jpeg/CUDA_JPEG/images/lena_std.tif");    
    cv::namedWindow("Image Window", cv::WINDOW_NORMAL);
    cv::imshow("Image Window", image);
    cv::waitKey(0);

    printf("\nSetting up the problem..."); fflush(stdout);
    // startTime(&timer);

    float *A_h, *B_h, *C_h;
    float *A_d, *B_d, *C_d;
    size_t mat_sz;
    unsigned matDim;
    dim3 dim_grid, dim_block;

    if (argc == 1) {
        matDim = 1000;
    } else if (argc == 2) {
        matDim = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
      "\n    Usage: ./mat-add                # All matrices are 1000 x 1000"
      "\n    Usage: ./mat-add <m>            # All matrices are m x m"
      "\n");
        exit(0);
    }
   
    mat_sz = matDim*matDim;

    A_h = (float*) malloc( sizeof(float)*mat_sz ); //returns a char pointer or void pointer for malloc
    //Since I want float cast to float
    //size of float * number of elements    
    
    for (unsigned int i=0; i < mat_sz; i++) { A_h[i] = (rand()%100)/100.00; }
    //creates an Array A_h on the host size of size mat_sz with elements 0->0.99
    B_h = (float*) malloc( sizeof(float)*mat_sz );
    for (unsigned int i=0; i < mat_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    C_h = (float*) malloc( sizeof(float)*mat_sz );

    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    A: %u x %u\n    B: %u x %u\n    C: %u x %u\n", matDim, matDim,
        matDim, matDim, matDim, matDim);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    // startTime(&timer);

    /*************************************************************************/    
    
    //allocate input vectors in device memory of size number of elements in mat_size * sizeof each element
    
    hipMalloc(&A_d, sizeof(float)*mat_sz);
    hipMalloc(&B_d, sizeof(float)*mat_sz);
    hipMalloc(&C_d, sizeof(float)*mat_sz);

    //INSERT CODE HERE

    /*************************************************************************/
    hipDeviceSynchronize();
    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------
    printf("Copying data from host to device..."); fflush(stdout);
    // startTime(&timer);
	
    /*************************************************************************/
    //INSERT CODE HERE

    hipMemcpy(A_d, A_h, sizeof(float)*mat_sz, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, sizeof(float)*mat_sz, hipMemcpyHostToDevice);
    hipMemcpy(C_d, C_h, sizeof(float)*mat_sz, hipMemcpyHostToDevice);

    /*************************************************************************/
    hipDeviceSynchronize();
    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel using standard mat-add interface ---------------------------
    printf("Launching kernel..."); fflush(stdout);
    // startTime(&timer);
    basicMatAdd(mat_sz, A_d, B_d, C_d);

    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");
    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    // startTime(&timer);

    /*************************************************************************/
    //INSERT CODE HERE
    
    //copy results from device memory to host
    hipMemcpy(A_h, A_d, sizeof(float)*mat_sz, hipMemcpyDeviceToHost);
    hipMemcpy(B_h, B_d, sizeof(float)*mat_sz, hipMemcpyDeviceToHost);
    hipMemcpy(C_h, C_d, sizeof(float)*mat_sz, hipMemcpyDeviceToHost);
    /*************************************************************************/
    hipDeviceSynchronize();
    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results...\n"); fflush(stdout);

    // verify(A_h, B_h, C_h, matDim);


    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);

    /*************************************************************************/
    //INSERT CODE HERE

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    cv::destroyWindow("Image Window");
    /*************************************************************************/
    return 0;
}

