#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <opencv2/opencv.hpp>
#define BLOCK_SIZE 8

using namespace cv;
using namespace std;

//DCT matrix T obtained from matlab dctmtx(8)
__constant__ float dctMatrix[BLOCK_SIZE * BLOCK_SIZE] = {
    0.3536, 0.3536, 0.3536, 0.3536, 0.3536, 0.3536, 0.3536, 0.3536,
    0.4904, 0.4157, 0.2778, 0.0975, -0.0975, -0.2778, -0.4157, -0.4904,
    0.4619, 0.1913, -0.1913, -0.4619, -0.4619, -0.1913, 0.1913, 0.4619,
    0.4157, -0.0975, -0.4904, -0.2778, 0.2778, 0.4904, 0.0975, -0.4157,
    0.3536, -0.3536, -0.3536, 0.3536, 0.3536, -0.3536, -0.3536, 0.3536,
    0.2778, -0.4904, 0.0975, 0.4157, -0.4157, -0.0975, 0.4904, -0.2778,
    0.1913, -0.4619, 0.4619, -0.1913, -0.1913, 0.4619, -0.4619, 0.1913,
    0.0975, -0.2778, 0.4157, -0.4904, 0.4904, -0.4157, 0.2778, -0.0975
};

//transposed DCT matrix T' obtained from matlab dctmtx(8) with a transpose
__constant__ float IdctMatrix[BLOCK_SIZE * BLOCK_SIZE] = {
    0.3536, 0.4904, 0.4619, 0.4157, 0.3536, 0.2778, 0.1913, 0.0975,
    0.3536, 0.4157, 0.1913, -0.0975, -0.3536, -0.4904, -0.4619, -0.2778,
    0.3536, 0.2778, -0.1913, -0.4904, -0.3536, 0.0975, 0.4619, 0.4157,
    0.3536, 0.0975, -0.4619, -0.2778, 0.3536, 0.4157, -0.1913, -0.4904,
    0.3536, -0.0975, -0.4619, 0.2778, 0.3536, -0.4157, -0.1913, 0.4904,
    0.3536, -0.2778, -0.1913, 0.4904, -0.3536, -0.0975, 0.4619, -0.4157,
    0.3536, -0.4157, 0.1913, 0.0975, -0.3536, 0.4904, -0.4619, 0.2778,
    0.3536, -0.4904, 0.4619, -0.4157, 0.3536, -0.2778, 0.1913, -0.0975
};

//F(p,q) = T * f(x,y) * T'

__global__ void DCT(int numRows, int numCols, float *d_image, float *DCT_res) {

    __shared__ float cache[BLOCK_SIZE*BLOCK_SIZE];  
    int y = threadIdx.y + (blockDim.y*blockIdx.y);
    int x = threadIdx.x + (blockDim.x*blockIdx.x);    

    float sum = 0.0f;

    if(y < numRows && x < numCols){
        cache[threadIdx.y*BLOCK_SIZE + threadIdx.x] = d_image[y*numCols + x];
        __syncthreads();

        
        for(int k = 0; k < BLOCK_SIZE; k++){
            sum += dctMatrix[threadIdx.y*BLOCK_SIZE + k] * cache[k*BLOCK_SIZE + threadIdx.x];
        }
        
        __syncthreads();
        DCT_res[y * numCols + x] = sum;
    }    

}

__global__ void IDCT(int numRows, int numCols, float *DCT_res, float *IDCT_res) {

    __shared__ float cache[BLOCK_SIZE*BLOCK_SIZE];  
    int y = threadIdx.y + (blockDim.y*blockIdx.y);
    int x = threadIdx.x + (blockDim.x*blockIdx.x);    

    float sum = 0.0f;

    if(y < numRows && x < numCols){
        cache[threadIdx.y*BLOCK_SIZE + threadIdx.x] = DCT_res[y*numCols + x];
        __syncthreads();
       
        for(int k = 0; k < BLOCK_SIZE; k++){
            sum += IdctMatrix[threadIdx.y*BLOCK_SIZE + k] * cache[k*BLOCK_SIZE + threadIdx.x];
        }

        __syncthreads();
        IDCT_res[y * numCols + x] = sum;
    }

}  

    //     __syncthreads();

    //     //intermediate result for 1D DCT need to now multiply by Transposed matrix
    //     cache[threadIdx.y*BLOCK_SIZE + threadIdx.x] = sum;

    //     __syncthreads();

    //     sum = 0.0f;

    //     for(int k = 0; k < BLOCK_SIZE; k++){
    //         sum += IdctMatrix[threadIdx.y * BLOCK_SIZE + k] * cache[k*BLOCK_SIZE + threadIdx.x];
    //     }

    //     __syncthreads();

    //     result[y * numCols + x] = sum;

    // }  


    


    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (dim x dim) matrix
     *   where B is a (dim x dim) matrix
     *   where C is a (dim x dim) matrix
     *
     ********************************************************************/


    /*************************************************************************/
    // INSERT KERNEL CODE HERE

    /*************************************************************************/









void LaunchDCT(const int row, const int col, float *d_image, float *DCT_res)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 blocksPerGrid(ceil(row/(float)threadsPerBlock.x), ceil(col/(float)threadsPerBlock.y), 1);
    DCT<<<blocksPerGrid, threadsPerBlock>>>(row, col, d_image, DCT_res);

}

void LaunchIDCT(const int row, const int col, float *DCT_res, float *IDCT_res)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 blocksPerGrid(ceil(row/(float)threadsPerBlock.x), ceil(col/(float)threadsPerBlock.y), 1);
    IDCT<<<blocksPerGrid, threadsPerBlock>>>(row, col, DCT_res, IDCT_res);

}

