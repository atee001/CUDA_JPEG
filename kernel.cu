#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/opencv.hpp>

#define TILE_SIZE 16

__global__ void matAdd(int dim, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (dim x dim) matrix
     *   where B is a (dim x dim) matrix
     *   where C is a (dim x dim) matrix
     *
     ********************************************************************/


    /*************************************************************************/
    // INSERT KERNEL CODE HERE

    /*************************************************************************/

}

void basicMatAdd(int dim, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------


}

