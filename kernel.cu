
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void matAdd(int dim, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (dim x dim) matrix
     *   where B is a (dim x dim) matrix
     *   where C is a (dim x dim) matrix
     *
     ********************************************************************/


    /*************************************************************************/
    // INSERT KERNEL CODE HERE

    /*************************************************************************/

}

void basicMatAdd(int dim, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------


}

